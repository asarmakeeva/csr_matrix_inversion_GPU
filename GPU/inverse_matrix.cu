#include "hip/hip_runtime.h"
#ifndef GPU_INVERSE_CU_
#define GPU_INVERSE_CU_

#include <iostream>
#include <hipblas.h>
#include <stdio.h>
#include <fstream>
#include <string.h>
#include <omp.h>

using namespace std;

#include "struct.cu"
//#include "kernel.cu"
#include "MatrixVectorProduct.cu"

const int warp=32;

void InverseMatrix(
		CUSMA::CSRmatrix A,
		double* x);

void SInverseMatrix(
		CUSMA::CSRmatrix A,
		double* x);

extern "C" void InverseMatrix(int* row_offsets, int* column_indices, double* values, 
                              double* inv_values, double* inv_values1, int* num_rows, int* num_elem)
{
    CUSMA::CSRmatrix A;
    A.dim = *num_rows;
    A.NumEl = *num_elem;
    A.V = values;
    A.NC = column_indices;
    A.NL = row_offsets;

    int maxDimGrid;
	
    // определение количества и размера блоков
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties (&deviceProp,0);
    int maxDimBlock = deviceProp.maxThreadsDim[0];		// максимальный размер блока
    maxDimGrid = deviceProp.maxGridSize[1];				// максимальный размер сетки

    cout << "Matrix inversion of the method with the completion" <<endl
	 << "Dimension of a matrix " << A.dim << " Nonzero Element = " << A.NumEl << endl
	 << "MaxDimGrid " << maxDimGrid << " MaxDimBlock " << maxDimBlock << endl;


double time1,time2,time3, time4;
cout<<"InverseMatrix:"<<endl;
time1 = omp_get_wtime();
  InverseMatrix(A, inv_values);
time2 = omp_get_wtime();
cout<<"Time = "<<time2-time1<<endl;
cout<<"SInverseMatrix"<<endl;
time3 = omp_get_wtime();
  SInverseMatrix(A, inv_values1);
time4 = omp_get_wtime();
cout<<"Time = "<<time4-time3<<endl;
        
}
void InverseMatrix(
/*			int* row_offsets,
			int* column_indices,
			double* values,
			double* x,
			double* b,
			int* num_rows,
			int* num_elem,
			int* num_iter,
			double* tol,
			int* prectype)
*/
		CUSMA::CSRmatrix A,
		double* x)
{
// hipDeviceReset();
 hipSetDevice(0);
	double *AV = A.V;//values;
	double *res = x;
	int* ANC = A.NC;//column_indices;
	int* ANL = A.NL;//row_offsets;
	int dim = A.dim;//*num_rows;
	int NumEl = A.NumEl;//*num_elem;
	// указатели на массивы в видеопамяти
	double *d_AV;//, *d_Inverse; 

	int *d_ANL; 
	int *d_ANC;
	int nwarp;
	int maxDimGrid, DimBlock, DimGrid;
	
	// определение количества и размера блоков
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties (&deviceProp,0);
	int maxDimBlock = deviceProp.maxThreadsDim[0];		// максимальный размер блока
	maxDimGrid = deviceProp.maxGridSize[1];				// максимальный размер сетки
/*	cout << "Matrix inversion of the method with the completion" <<endl
		 << "Dimension of a matrix " << dim << " Nonzero Element = " << NumEl << endl
		 << "MaxDimGrid " << maxDimGrid << " MaxDimBlock " << maxDimBlock << endl;
*/
	nwarp = 1 + dim/((maxDimGrid-1)*warp);
	DimBlock = nwarp * warp;
	DimGrid = dim / DimBlock +1;
	// установка количества блоков
	dim3 grid(DimGrid, 1, 1);
	// установка количества потоков в блоке
	dim3 threads(DimBlock, 1, 1);
	// выделение видеопамяти
	hipMalloc((void **)&d_AV, sizeof(double)*NumEl);
//	hipMalloc((void **)&d_Inverse, sizeof(double)*dim*dim);
	hipMalloc((void **)&d_ANL, sizeof(int)*(dim+1));
	hipMalloc((void **)&d_ANC, sizeof(int)*NumEl);
	
	// копирование из оперативной памяти в видеопамять
	hipMemcpy(d_AV, AV, sizeof(double)*NumEl, hipMemcpyHostToDevice);
	hipMemcpy(d_ANL, ANL, sizeof(int)*(dim+1), hipMemcpyHostToDevice);
	hipMemcpy(d_ANC, ANC, sizeof(int)*NumEl, hipMemcpyHostToDevice);

//	cout << "Memory allocation time - " << t0/1000 << endl;	// время выделения памяти GPU

	double *diagA;			// диагональный предобуславлеватель
	double *ak, *v;
	hipMalloc((void **)&ak, sizeof(double)*dim);
	hipMalloc((void **)&v, sizeof(double)*dim);
//	cout << "Time calculation diagonal preconditioner - " << t1 << endl;
	// время вычисления диагонального предобуславлевателя

	int k=0;
// **************************************** Ershow met
        unsigned int i,j;
	double tim1,tim2,result1,buf1,tim3,tim4,result2,buf2;
        ofstream yyy;
        
        double **d_Inverse;
        d_Inverse = new double* [dim];
            
        for(i=0 ; i<dim ; i++)
        {
            hipMalloc((void **)&d_Inverse[i], sizeof(double)*dim);
            iden_vec <<<grid, threads >>> (d_Inverse[i], dim, i);			// обнуление вектора результата
        }
        double *loc = new double [dim];
/*        yyy.open("MatE");
        for(i=0;i<dim;i++)
        { 	
            hipMemcpy(loc, d_Inverse[i], sizeof(double)*dim, hipMemcpyDeviceToHost);

            for(j=0;j<dim;j++)
                yyy<<loc[j]<<"   ";
            yyy<<endl;
        }
        yyy.close();
*/
//	hipblasHandle_t handle;
//	hipblasCreate(&handle);   
     
/*	hipMemcpy(res, d_Inverse, sizeof(double)*dim*dim, hipMemcpyDeviceToHost);

        yyy.open("MatE");
        for(i=0;i<dim*dim;i++)
            yyy<<res[i]<<endl;
        yyy.close();
*/

        double alpha,alpha2,alpha1;
	
        double *loc1 = new double [dim];
        double *loc2 = new double [dim];

//        yyy.open("Vectors");
        for(i=0;i<dim;i++)
        {
            vec_def <<<grid, threads >>> (v , dim);
            SetV <<<grid, threads >>> (v, d_AV, d_ANC, ANL[i+1]-ANL[i], ANL[i], i);
            hipblasDcopy(dim, d_Inverse[i], 1, ak, 1);
            for(j=0 ; j<dim ; j++)
            {
//                hipMemcpy(loc, v, sizeof(double)*dim, hipMemcpyDeviceToHost);
//                hipMemcpy(loc1, ak, sizeof(double)*dim, hipMemcpyDeviceToHost);
//                hipMemcpy(loc2, d_Inverse[j], sizeof(double)*dim, hipMemcpyDeviceToHost);
//                for(unsigned k=0;k<dim;k++)
//                   yyy<<loc[k]<<"   "<<loc1[k]<<"   "<<loc2[k]<<endl;
//                yyy<<"***"<<endl;
        tim1 = omp_get_wtime();
	alpha1=hipblasDdot(dim, v, 1, d_Inverse[j], 1);
	tim2 = omp_get_wtime();
	buf1=tim2-tim1;
	result1+=buf1;

	tim3 = omp_get_wtime();
	alpha2=1+hipblasDdot(dim, v, 1, ak, 1);
	tim4 = omp_get_wtime();
	buf2=tim4-tim3;
	result2+=buf2;

        alpha =alpha1/alpha2;
                hipblasDaxpy(dim, -alpha, ak, 1, d_Inverse[j], 1);
            }
//            yyy<<"--------------------------"<<endl;
        
        }
//        yyy.close();
	yyy.open("time_vect_ershov.txt",ios::app);
	yyy<<"razmernost="<<dim<<endl;
	yyy<<"vectprod 1="<<result1<<endl;
	yyy<<"vectprod 2="<<result2<<endl;
	yyy<<"END"<<endl;
	yyy.close();
        maxDimGrid = deviceProp.maxGridSize[1];
        yyy.open("E");
          for(i=0;i<dim;i++)
        {
            MatrVectMul(maxDimGrid, v, d_Inverse[i], d_AV, d_ANC, d_ANL, dim, NumEl);
            hipMemcpy(loc, v, sizeof(double)*dim, hipMemcpyDeviceToHost);

            for(j=0;j<dim;j++)
                yyy<<loc[j]<<"   ";
            yyy<<endl;   
        }
        yyy.close();
	// копирование результата из видеопамяти в оперативную память
	for(i=0;i<dim;i++)
            hipMemcpy(res+dim*i, d_Inverse[i], sizeof(double)*dim, hipMemcpyDeviceToHost);
// *******************************************
//	cout << "Quentity of iterations " << k <<" Error: "<<nrmr/nrmb<< endl
//		 << "Time execution of the conjugate gradients method - " << t2/1000 << endl
// время решения системы методом сопряженных градиентов
//		 << "Total time - " << t/1000 << endl;
// общее время работы функции CG
//  освобождение памяти

        delete [] loc1;
        delete [] loc2;
        delete [] loc;
        for (i=0;i<dim;i++)
        hipFree(d_Inverse[i]);

	delete [] d_Inverse;
	hipFree(d_AV);
        hipFree(d_ANL);
	hipFree(d_ANC);
	hipFree(ak);
	hipFree(v);
//	hipblasDestroy(handle);
	
	hipDeviceReset();
}
// end InverseMatrix
// *********************************************

void MatrVectMul (const int maxDimGrid, 
	      double* Ax , 
	  const double *p, 
	  const int *d_ANL, 
	  const int dim,
	  const int NumEl)
{
    
    const size_t dimBlock = 128;                                              //количество нитей в блоке
    
    int nnz_per_row = NumEl / dim;                                            //среднее количество ненулевых элементов в строке
    unsigned int thr_per_vec;                                                 //количество нитей для вычисления одной координаты вектора
    
    if (nnz_per_row <=  2) thr_per_vec=2;
    else
    if (nnz_per_row <=  4)  thr_per_vec=4;
    else
    if (nnz_per_row <=  8)  thr_per_vec=8;
    else
    if (nnz_per_row <=  16)  thr_per_vec=16;
    else
    thr_per_vec=32;
    
    const size_t VECTORS_PER_BLOCK = dimBlock / thr_per_vec;         // количество векторов в блоке
    
    const size_t DimGrid = std::min<int>(maxDimGrid, (dim + VECTORS_PER_BLOCK-1)/VECTORS_PER_BLOCK); //количество блоков
    
    hipBindTexture(0, tex_b, p, sizeof(double)*dim);                                                   //"Привязка текстуры"
    
    dev_MatrVectMul <<<DimGrid, dimBlock>>> (Ax ,  d_ANL, dim, VECTORS_PER_BLOCK, thr_per_vec);  //выхов функции на девайсе
        //dev_MatrVectMul < VECTORS_PER_BLOCK, THREADS_PER_VECTOR > <<<DimGrid, dimBlock>>> (Ax ,  d_ANL, dim);
    hipUnbindTexture(tex_b);                                                                           //"Отвязка текстуры"
}


void CG(int* d_ANL,
    double* d_b, 
    int dim, 
    int NumEl, 
    int iter, 
    double acc,
    double *diagA,
    double *r,
    double *Ax,
    double *p,
    double *z,
    double *d_res,
    dim3 grid,
    dim3 threads,
    int maxDimGrid,
    int *k,
    double *buf1,double *buf2,double *buf3, double *buf4)
{
//cout<<"check1"<<endl;

    /*hipblasHandle_t handle;
    hipblasCreate(&handle);*/
    double    buff1,buff2,buff3,buff4;
    double ro1, ro0, alpha, alpha1, beta, nrmr, nrmb;
    int kk=0;buff1=0;buff2=0;buff3=0;buff4=0;
    double time1, time2,time3,time4,time5,time6,time7,time8,result1,result2,result3,result4;

//****************************************
//метод сопряжённых градиентов

/*    double *local = new double [dim];
    for(int j=0;j<dim;j++)
        local[j] = 0;
    local[number]=1;
    

    double *d_b;
    hipMalloc((void **)&d_b, sizeof(double)*dim);

    hipMemcpy(d_b, local, sizeof(double)*dim, hipMemcpyHostToDevice);

    hipMemcpy(local, d_b, sizeof(double)*dim, hipMemcpyDeviceToHost);
    for(int j=0;j<dim;j++)
        cout<<local[j]<<"   ";
    cout<<endl;*/

    hipblasDcopy(dim,d_b,1,r,1);                              // r = d_b

    double nrmb1, nrmr1;
    nrmb1 = hipblasDdot(dim,d_b,1,d_b,1);                      // nrmb1 = (d_b, d_b)
    nrmb=sqrt(nrmb1);
    nrmr=nrmb;
//cout<<"check2"<<endl;

    DiagVectMul<<< grid, threads >>>(z ,diagA, r, dim);             // z = M*r, M - диагональный предобуславлеватель
//cout<<"check3"<<endl;
    hipblasDcopy (dim,z,1,p,1);                               // p = z
    ro1 = hipblasDdot (dim,z,1,r,1);                           //ro1 = (z,r)
//    cout<<"nrmb = "<<nrmb<<endl;
//cout<<"check4"<<endl;

    if (nrmb)
    while ( ((nrmr/nrmb) > acc) && (kk <= iter ) )
    {
	time1 = omp_get_wtime();
	MatrVectMul(maxDimGrid, Ax , p, d_ANL, dim, NumEl);	 //матрично-векторное произведение  Ax = A * p
	time2 = omp_get_wtime();
	result1=time2-time1;
	buff1+=result1;

	alpha1=0;

	time3 = omp_get_wtime();
	alpha1 = hipblasDdot(dim,p,1,Ax,1);                 //alpha1 = (p, Ax)
	time4 = omp_get_wtime();
	result2=time4-time3;
	buff2+=result2;

	alpha=ro1 / alpha1;		
	hipblasDaxpy(dim,alpha,p,1,d_res,1) ;             //d_res = d_res + alpha * p
	alpha1=-alpha;
	hipblasDaxpy(dim,alpha1,Ax,1,r,1) ;               //r = r - alpha * Ax
	
	time5 = omp_get_wtime();
	nrmr1 = hipblasDdot(dim,r,1,r,1) ;                  // nrmr1 = (r,r)
	time6 = omp_get_wtime();
	result3=time6-time5;
	buff3+=result3;

	nrmr=sqrt(nrmr1);
	DiagVectMul<<< grid, threads>>>(z ,diagA, r, dim);      //z = M*r, M - диагональный предобуславлеватель
	ro0 = ro1 ;

	time7 = omp_get_wtime();
	ro1 = hipblasDdot (dim , z , 1 , r , 1 );          //ro1 = (z,r)
	time8 = omp_get_wtime();
	result4=time8-time7;
	buff4+=result4;

	beta=ro1 / ro0 ;
	hipblasDscal (dim , beta , p , 1 ) ;             //p = beta*p
	alpha1=1.0;
	hipblasDaxpy (dim ,alpha1 , z , 1 , p , 1 ) ;    //p = p + z
	kk++;
    }
//конец метода сопряжённых градиентов
//*******************************************
//cout<<"check5"<<endl;
*k=kk;
*buf1=buff1;
*buf3=buff3;
*buf2=buff2;
*buf4=buff4;

// освобождение памяти
//	hipblasDestroy(handle);
}


void SInverseMatrix(
		CUSMA::CSRmatrix A,
		double* x)
{
// hipDeviceReset();
 hipSetDevice(0);
	double *AV = A.V;//values;
	double *res = x;
	int* ANC = A.NC;//column_indices;
	int* ANL = A.NL;//row_offsets;
	int dim = A.dim;//*num_rows;
	int NumEl = A.NumEl;//*num_elem;
	
	long Norma_A=0,Sum=0;
	for(int i=0;i<A.dim;i++)
	{
	    for(int j=A.NL[i]; j<A.NL[i+1]; j++)
	    {
		Sum+=abs(A.V[j]) * abs(A.V[j]);
	    }
	}
	Norma_A = sqrt(Sum);

	// указатели на массивы в видеопамяти
	double *d_AV;//, *d_Inverse; 
	int *d_ANL; 
	int *d_ANC;
	int nwarp;
	int maxDimGrid, DimBlock, DimGrid;
	
	// определение количества и размера блоков
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties (&deviceProp,0);
	int maxDimBlock = deviceProp.maxThreadsDim[0];		// максимальный размер блока
	maxDimGrid = deviceProp.maxGridSize[1];				// максимальный размер сетки

/*	cout << "Matrix inversion of the method with the completion" <<endl
		 << "Dimension of a matrix " << dim << " Nonzero Element = " << NumEl << endl
		 << "MaxDimGrid " << maxDimGrid << " MaxDimBlock " << maxDimBlock << endl;
*/
	nwarp = 1 + dim/((maxDimGrid-1)*warp);
	DimBlock = nwarp * warp;
	DimGrid = dim / DimBlock +1;
	
	// установка количества блоков
	dim3 grid(DimGrid, 1, 1);
	// установка количества потоков в блоке
	dim3 threads(DimBlock, 1, 1);
	// выделение видеопамяти
	hipMalloc((void **)&d_AV, sizeof(double)*NumEl);
//	hipMalloc((void **)&d_Inverse, sizeof(double)*dim*dim);
	hipMalloc((void **)&d_ANL, sizeof(int)*(dim+1));
	hipMalloc((void **)&d_ANC, sizeof(int)*NumEl);
	
	// копирование из оперативной памяти в видеопамять
	hipMemcpy(d_AV, AV, sizeof(double)*NumEl, hipMemcpyHostToDevice);
	hipMemcpy(d_ANL, ANL, sizeof(int)*(dim+1), hipMemcpyHostToDevice);
	hipMemcpy(d_ANC, ANC, sizeof(int)*NumEl, hipMemcpyHostToDevice);

        hipBindTexture(0, tex_AV, d_AV, sizeof(double)*NumEl);
        hipBindTexture(0, tex_ANC, d_ANC, sizeof(int)*NumEl);

//	cout << "Memory allocation time - " << t0/1000 << endl;	// время выделения памяти GPU

//	cout << "Time calculation diagonal preconditioner - " << t1 << endl;	// время вычисления диагонального предобуславлевателя

	//int k=0;
// ****************************************CG metod
        unsigned int i,j;
	int k,buf_k;
        ofstream yyy;
        
        double **d_Inverse;
        d_Inverse = new double* [dim];
            
        for(i=0 ; i<dim ; i++)
        {
            hipMalloc((void **)&d_Inverse[i], sizeof(double)*dim);
            vec_def <<<grid, threads >>> (d_Inverse[i], dim);			// обнуление вектора результата
        }

/*        yyy.open("MatE");
        for(i=0;i<dim;i++)
        { 	
            hipMemcpy(loc, d_Inverse[i], sizeof(double)*dim, hipMemcpyDeviceToHost);

            for(j=0;j<dim;j++)
                yyy<<loc[j]<<"   ";
            yyy<<endl;
        }
        yyy.close();
*/
//	hipblasHandle_t handle;
//	hipblasCreate(&handle);   
     
/*	hipMemcpy(res, d_Inverse, sizeof(double)*dim*dim, hipMemcpyDeviceToHost);

        yyy.open("MatE");
        for(i=0;i<dim*dim;i++)
            yyy<<res[i]<<endl;
        yyy.close();
*/

	double *diagA;			// диагональный предобуславлеватель
        double *r, *Ax, *p, *z, *d_b;
        hipMalloc((void **)&r, sizeof(double)*dim);
        hipMalloc((void **)&Ax, sizeof(double)*dim);
        hipMalloc((void **)&p, sizeof(double)*dim);
        hipMalloc((void **)&z, sizeof(double)*dim);
        hipMalloc((void **)&diagA, sizeof(double)*dim);
        hipMalloc((void **)&d_b, sizeof(double)*dim);

	double buf1,buf2,buf3,buf4,tim1,tim2,tim3,tim4;
    k=0;buf1=0;buf2=0;buf3=0;buf4=0;
        diag<<< grid, threads >>>(diagA, d_ANL, dim);                   //вычисление диагонального предобуславлевателя


        double acc = 1e-08;
//        double *local = new double [dim];
        for(i=0;i<dim;i++)
        {
//	    hipMemcpy(d_b, local, sizeof(double)*dim, hipMemcpyHostToDevice);
            iden_vec <<<grid, threads >>> (d_b, dim, i);			//identifix ed vect
//            line_def <<<grid, threads, 0, stream >>> (d_Inverse[i], dim);
	    CG(d_ANL, d_b, dim, NumEl, dim, acc, diagA, r, Ax, p, z, d_Inverse[i], grid, threads, maxDimGrid,&k,&buf1,&buf2,&buf3,&buf4);
            tim1+=buf1;
	    tim2+=buf2;
	    tim3+=buf3;
	    tim4+=buf4;
	    buf_k+=k;
        }

ofstream print;
print.open("time_CG.txt",ios::app);
print<<"razmernost="<<dim<<endl;
print<<"time for matrix vect mult="<<tim1<<endl;
print<<"time for vectprod ="<<tim2<<endl;
print<<"time for vectprod ="<<tim3<<endl;
print<<"time for vectprod ="<<tim4<<endl;
print<<"END"<<endl;
print.close();
//delete [] local;

        maxDimGrid = deviceProp.maxGridSize[1];	

        double *loc = new double [dim];

        yyy.open("SE");

        for(i=0;i<dim;i++)
        {
            MatrVectMul(maxDimGrid, r, d_Inverse[i], d_AV, d_ANC, d_ANL, dim, NumEl);
            hipMemcpy(loc, r, sizeof(double)*dim, hipMemcpyDeviceToHost);

            for(j=0;j<dim;j++)
                yyy<<loc[j]<<"   ";
            yyy<<endl;   
        }
                  
        yyy.close();
ofstream write;
write.open("iter.txt",ios::app);
write<<"razmernost="<<dim<<endl;
write<<buf_k<<endl;
write<<"END"<<endl;
write.close();
                                                                                                                                                                                                                                                                
	// копирование результата из видеопамяти в оперативную память
	for(i=0;i<dim;i++)
            hipMemcpy(res+dim*i, d_Inverse[i], sizeof(double)*dim, hipMemcpyDeviceToHost);
// *******************************************
//	cout << "Quentity of iterations " << k <<" Error: "<<nrmr/nrmb<< endl
//		 << "Time execution of the conjugate gradients method - " << t2/1000 << endl	// время решения системы методом сопряженных градиентов
//		 << "Total time - " << t/1000 << endl;											// общее время работы функции CG
	//  освобождение памяти
        hipUnbindTexture(tex_AV);
        hipUnbindTexture(tex_ANC);
	hipFree(d_AV);
	for(i=0;i<dim;i++)
            hipFree(d_Inverse[i]);
        delete [] d_Inverse;
	hipFree(d_ANL);
	hipFree(d_ANC);
	hipFree(r);
	hipFree(Ax);
	hipFree(p);
	hipFree(z);
	hipFree(diagA);
	hipFree(d_b);

//	hipblasDestroy(handle);
	
	hipDeviceReset();
/*
	long Norma_invA=0;
	Sum = 0;
	for(int i=0;i<dim;i++)
	{
	    for(int j=0; j<dim; j++)
	    {
		if (i==j) res[j+dim*i]-=1;
		Sum+=abs(res[j+dim*i]) * abs(res[j+dim*i]);
	    }
	}
	Norma_invA = sqrt(Sum);
*/
	//cout<<"Obuslovl. A = "<<Norma_A * Norma_invA<<" norma A= "<<Norma_A<<;
//	cout<<" norma INV ="<<Norma_invA<<endl;
}
// end SInverseMatrix
// *********************************************


#endif //GPU_INVERSE_CU_
